#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include "helper.h"


__global__ static void write_id(int* mem, int id)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        mem[0] = id;
    }
}

int main(int argc, char** argv)
{
    std::vector<int> gpu_ids{0, 0, 0};
    // std::vector<int> gpu_ids{0, 1, 2};

    std::vector<int*>            d_buf(gpu_ids.size());
    std::vector<int>             h_buf(2 * gpu_ids.size());
    std::vector<hipStream_t>    streams(gpu_ids.size());
    std::vector<hipGraph_t>     graphs(gpu_ids.size());
    std::vector<hipGraphExec_t> exec_graphs(gpu_ids.size());
    std::vector<hipGraphNode_t> kernel_nodes(gpu_ids.size());
    std::vector<hipGraphNode_t> event_nodes(gpu_ids.size());
    std::vector<hipEvent_t>     events(gpu_ids.size());

    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        CUDA_ERROR(hipMalloc((void**)gpu_ids[i], sizeof(int)));
        CUDA_ERROR(hipGraphCreate(&graphs[i], 0));
        CUDA_ERROR(hipStreamCreate(&streams[i]));
    }


    // 1st node is to write the GPU id to d_buf
    // 2nd node is a record event after the 1st node
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));

        // 1st
        void*                kernelArgs[2] = {&d_buf[i], &gpu_ids[i]};
        hipKernelNodeParams kernelNodeParams = {0};
        kernelNodeParams.func = (void*)write_id;
        kernelNodeParams.gridDim = dim3(1, 1, 1);
        kernelNodeParams.blockDim = dim3(1, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams = (void**)kernelArgs;
        kernelNodeParams.extra = NULL;

        CUDA_ERROR(hipGraphAddKernelNode(&kernel_nodes[i], graphs[i], NULL, 0,
                                          &kernelNodeParams));

        // 2nd
        CUDA_ERROR(hipEventCreate(&events[i]));
        CUDA_ERROR(hipGraphAddEventRecordNode(&event_nodes[i], graphs[i],
                                               &kernel_nodes[i], 1, events[i]));
    }

    // 3rd nodes are the wait node on the event nodes from the two other graphs
    // 4th nodes are the memcpy from device to host where we copy the other two
    // graph data
    for (size_t i = 0; i < gpu_ids.size(); ++i) {

        std::vector<hipGraphNode_t> wait_nodes(2);

        size_t i_next = (i + 1) % gpu_ids.size();
        size_t i_prev = (i == 0) ? gpu_ids.size() - 1 : i - 1;

        // 3rd
        CUDA_ERROR(hipGraphAddEventWaitNode(
            &wait_nodes[0], graphs[i], &event_nodes[i], 1, events[i_next]));

        CUDA_ERROR(hipGraphAddEventWaitNode(
            &wait_nodes[1], graphs[i], &event_nodes[i], 1, events[i_prev]));

        // 4th
        hipGraphNode_t n_next, n_prev;
        CUDA_ERROR(hipGraphAddMemcpyNode1D(
            &n_next, graphs[i], wait_nodes.data(), wait_nodes.size(),
            &h_buf[i * 2], d_buf[i_next], sizeof(int), hipMemcpyDeviceToHost));

        CUDA_ERROR(hipGraphAddMemcpyNode1D(
            &n_prev, graphs[i], wait_nodes.data(), wait_nodes.size(),
            &h_buf[i * 2 + 1], d_buf[i_prev], sizeof(int),
            hipMemcpyDeviceToHost));


        // make the graph executable
        hipGraphNode_t pErrorNode = nullptr;
        const size_t    bufferSize = 1024;
        char            pLogBuffer[bufferSize];
        hipError_t     res = ::hipGraphInstantiate(
            &exec_graphs[i], graphs[i], &pErrorNode, pLogBuffer, bufferSize);
        bool trucatedErrorMessage = (pLogBuffer[bufferSize - 1] == '\0');
        pLogBuffer[bufferSize - 1] = '\0';
        if (res != hipSuccess) {
            std::cout << "\n\t Error: " << hipGetErrorString(res);
            std::cout << "\n\t Error: " << pLogBuffer;
            std::cout << "\n\t Error: Related Graph Node ->"
                      << reinterpret_cast<char*>(pErrorNode);
            if (trucatedErrorMessage) {
                std::cout << "\n Error: previous error message was truncated";
            }
        }
    }


    // launch the graphs
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipGraphLaunch(exec_graphs[i], streams[i]));
    }


    // sync and check the output
    for (int i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        CUDA_ERROR(hipDeviceSynchronize());

        int i_next = (i + 1) % gpu_ids.size();
        int i_prev = (i == 0) ? gpu_ids.size() - 1 : i - 1;

        printf("\n*** ID = %d\n", i);
        printf("\n i_next = %d, buf= %d", i_next, h_buf[2 * i]);
        printf("\n i_prev = %d, buf= %d\n", i_prev, h_buf[2 * i + 1]);
    }


    // clean up
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipGraphExecDestroy(exec_graphs[i]));
        CUDA_ERROR(hipGraphDestroy(graphs[i]));
        CUDA_ERROR(hipStreamDestroy(streams[i]));
        CUDA_ERROR(hipFree(d_buf[i]));
        CUDA_ERROR(hipEventDestroy(events[i]));
    }
}
