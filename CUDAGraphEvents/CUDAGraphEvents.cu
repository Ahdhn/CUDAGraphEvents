#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include "helper.h"


__global__ static void write_id(int* mem, int id)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        mem[0] = id;
    }
}

int main(int argc, char** argv)
{
    std::vector<int> gpu_ids{0, 0, 0};
    // std::vector<int> gpu_ids{0, 1, 2};

    std::vector<int> values{11, 22, 33};

    std::vector<int*>            d_buf(gpu_ids.size());
    std::vector<int>             h_buf(2 * gpu_ids.size());
    std::vector<hipStream_t>    streams(gpu_ids.size());
    std::vector<hipGraph_t>     graphs(gpu_ids.size());
    std::vector<hipGraphExec_t> exec_graphs(gpu_ids.size());
    std::vector<hipGraphNode_t> kernel_nodes(gpu_ids.size());
    std::vector<hipGraphNode_t> event_nodes(gpu_ids.size());
    std::vector<hipEvent_t>     events(gpu_ids.size());

    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        int* buf = NULL;
        CUDA_ERROR(hipMalloc((void**)&buf, sizeof(int)));
        d_buf[i] = buf;
        CUDA_ERROR(hipGraphCreate(&graphs[i], 0));
        CUDA_ERROR(hipStreamCreate(&streams[i]));
        CUDA_ERROR(hipEventCreate(&events[i]));
    }


    // 1st node is to write the GPU id to d_buf
    // 2nd node is a record event after the 1st node
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));

        // 1st
        void*                kernelArgs[2] = {&d_buf[i], &values[i]};
        hipKernelNodeParams kernelNodeParams = {0};
        kernelNodeParams.func = (void*)write_id;
        kernelNodeParams.gridDim = dim3(1, 1, 1);
        kernelNodeParams.blockDim = dim3(1, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams = (void**)kernelArgs;
        kernelNodeParams.extra = NULL;

        CUDA_ERROR(hipGraphAddKernelNode(&kernel_nodes[i], graphs[i], NULL, 0,
                                          &kernelNodeParams));

        // 2nd
        CUDA_ERROR(hipGraphAddEventRecordNode(&event_nodes[i], graphs[i],
                                               &kernel_nodes[i], 1, events[i]));
    }

    // 3rd nodes are the wait node on the event nodes from the two other graphs
    // 4th nodes are the memcpy from device to host where we copy the other two
    // graph data
    for (size_t i = 0; i < gpu_ids.size(); ++i) {

        std::vector<hipGraphNode_t> wait_nodes(2);

        size_t i_next = (i + 1) % gpu_ids.size();
        size_t i_prev = (i == 0) ? gpu_ids.size() - 1 : i - 1;

        // 3rd
        CUDA_ERROR(hipGraphAddEventWaitNode(
            &wait_nodes[0], graphs[i], &event_nodes[i], 1, events[i_next]));

        CUDA_ERROR(hipGraphAddEventWaitNode(
            &wait_nodes[1], graphs[i], &event_nodes[i], 1, events[i_prev]));

        // 4th
        hipGraphNode_t n_next, n_prev;
        CUDA_ERROR(hipGraphAddMemcpyNode1D(
            &n_next, graphs[i], wait_nodes.data(), wait_nodes.size(),
            &h_buf[i * 2], d_buf[i_next], sizeof(int), hipMemcpyDeviceToHost));

        CUDA_ERROR(hipGraphAddMemcpyNode1D(
            &n_prev, graphs[i], wait_nodes.data(), wait_nodes.size(),
            &h_buf[i * 2 + 1], d_buf[i_prev], sizeof(int),
            hipMemcpyDeviceToHost));


        // make the graph executable
        hipGraphNode_t pErrorNode = nullptr;
        const size_t    bufferSize = 1024;
        char            pLogBuffer[bufferSize];
        hipError_t     res = ::hipGraphInstantiate(
            &exec_graphs[i], graphs[i], &pErrorNode, pLogBuffer, bufferSize);
        bool trucatedErrorMessage = (pLogBuffer[bufferSize - 1] == '\0');
        pLogBuffer[bufferSize - 1] = '\0';
        if (res != hipSuccess) {
            std::cout << "\n Error: " << hipGetErrorString(res);
            std::cout << "\n Error: " << pLogBuffer;
            std::cout << "\n Error: Related Graph Node ->"
                      << reinterpret_cast<char*>(pErrorNode);
            if (trucatedErrorMessage) {
                std::cout << "\n Error: previous error message was truncated";
            }
        }
    }


// launch the graphs
#pragma omp parallel for num_threads(gpu_ids.size())
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipGraphLaunch(exec_graphs[i], streams[i]));
        CUDA_ERROR(hipStreamSynchronize(streams[i]));
    }


    // check the output
    for (int i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        CUDA_ERROR(hipDeviceSynchronize());
    }


    // copy the ground truth to host
    std::vector<int> truth(gpu_ids.size());
    for (int i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        CUDA_ERROR(hipMemcpy(&truth[i], d_buf[i], sizeof(int),
                              hipMemcpyDeviceToHost));
    }

    // sync and check the output
    for (int i = 0; i < gpu_ids.size(); ++i) {
        int i_next = (i + 1) % gpu_ids.size();
        int i_prev = (i == 0) ? gpu_ids.size() - 1 : i - 1;

        printf("\n*** ID = %d\n", i);
        printf("\n i_next = %d, val= %d, truth= %d, buf= %d", i_next,
               values[i_next], truth[i_next], h_buf[2 * i]);

        printf("\n i_prev = %d, val= %d, truth= %d, buf= %d\n", i_prev,
               values[i_prev], truth[i_prev], h_buf[2 * i + 1]);
    }


    // clean up
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipGraphExecDestroy(exec_graphs[i]));
        CUDA_ERROR(hipGraphDestroy(graphs[i]));
        CUDA_ERROR(hipStreamDestroy(streams[i]));
        CUDA_ERROR(hipFree(d_buf[i]));
        CUDA_ERROR(hipEventDestroy(events[i]));
    }
}
