#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include "helper.h"


__global__ static void write_value(int* mem, int value)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        mem[0] = value;
    }
}

int main(int argc, char** argv)
{
    std::vector<int> gpu_ids{0, 0, 0};
    // std::vector<int> gpu_ids{0, 1, 2};

    std::vector<int> values{11, 22, 33};

    std::vector<int*>            d_buf(gpu_ids.size());
    std::vector<int>             h_result(2 * gpu_ids.size(), 99);
    std::vector<hipStream_t>    streams(gpu_ids.size());
    std::vector<hipGraph_t>     graphs(gpu_ids.size());
    std::vector<hipGraphExec_t> exec_graphs(gpu_ids.size());
    std::vector<hipGraphNode_t> kernel_nodes(gpu_ids.size());
    std::vector<hipGraphNode_t> event_nodes(gpu_ids.size());
    std::vector<hipEvent_t>     events(gpu_ids.size());

    // allocate device buffer and create graph, streams, and events
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        int* buf = NULL;
        CUDA_ERROR(hipMalloc((void**)&buf, sizeof(int)));
        d_buf[i] = buf;
        CUDA_ERROR(hipGraphCreate(&graphs[i], 0));
        CUDA_ERROR(hipStreamCreate(&streams[i]));
        CUDA_ERROR(hipEventCreate(&events[i]));
    }

    // initialize the device buffer with something
    for (int i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        CUDA_ERROR(hipMemcpy(d_buf[i], &h_result[i], sizeof(int),
                              hipMemcpyHostToDevice));
    }

    // 1st node is to write the GPU id to d_buf
    // 2nd node is a record event after the 1st node
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));

        // 1st
        void*                kernelArgs[2] = {&d_buf[i], &values[i]};
        hipKernelNodeParams kernelNodeParams = {0};
        kernelNodeParams.func = (void*)write_value;
        kernelNodeParams.gridDim = dim3(1, 1, 1);
        kernelNodeParams.blockDim = dim3(1, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams = (void**)kernelArgs;
        kernelNodeParams.extra = NULL;

        CUDA_ERROR(hipGraphAddKernelNode(&kernel_nodes[i], graphs[i], NULL, 0,
                                          &kernelNodeParams));

        // 2nd
        CUDA_ERROR(hipGraphAddEventRecordNode(&event_nodes[i], graphs[i],
                                               &kernel_nodes[i], 1, events[i]));
    }

    // 3rd nodes are the wait node on the event nodes from the two other graphs
    // 4th nodes are the memcpy from device to host where we copy the other two
    // graph data
    for (size_t i = 0; i < gpu_ids.size(); ++i) {

        std::vector<hipGraphNode_t> wait_nodes(2);

        size_t i_next = (i + 1) % gpu_ids.size();
        size_t i_prev = (i + gpu_ids.size() - 1) % gpu_ids.size();

        // 3rd
        CUDA_ERROR(hipGraphAddEventWaitNode(
            &wait_nodes[0], graphs[i], &event_nodes[i], 1, events[i_next]));

        CUDA_ERROR(hipGraphAddEventWaitNode(
            &wait_nodes[1], graphs[i], &event_nodes[i], 1, events[i_prev]));

        // 4th
        hipGraphNode_t n_next, n_prev;
        CUDA_ERROR(hipGraphAddMemcpyNode1D(
            &n_next, graphs[i], wait_nodes.data(), wait_nodes.size(),
            &h_result[i * 2], d_buf[i_next], sizeof(int),
            hipMemcpyDeviceToHost));

        CUDA_ERROR(hipGraphAddMemcpyNode1D(
            &n_prev, graphs[i], wait_nodes.data(), wait_nodes.size(),
            &h_result[i * 2 + 1], d_buf[i_prev], sizeof(int),
            hipMemcpyDeviceToHost));


        // make the graph executable and check for errors
        hipGraphNode_t pErrorNode = nullptr;
        const size_t    bufferSize = 1024;
        char            pLogBuffer[bufferSize];
        hipError_t     res = ::hipGraphInstantiate(
            &exec_graphs[i], graphs[i], &pErrorNode, pLogBuffer, bufferSize);
        bool trucatedErrorMessage = (pLogBuffer[bufferSize - 1] == '\0');
        pLogBuffer[bufferSize - 1] = '\0';
        if (res != hipSuccess) {
            std::cout << "\n Error: " << hipGetErrorString(res);
            std::cout << "\n Error: " << pLogBuffer;
            std::cout << "\n Error: Related Graph Node ->"
                      << reinterpret_cast<char*>(pErrorNode);
            if (trucatedErrorMessage) {
                std::cout << "\n Error: previous error message was truncated";
            }
        }
    }


// launch the graphs
#pragma omp parallel for num_threads(gpu_ids.size())
    for (int i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipGraphLaunch(exec_graphs[i], streams[i]));
        CUDA_ERROR(hipStreamSynchronize(streams[i]));
    }


    // copy the ground truth (values written to the device buffer) to the host
    std::vector<int> truth(gpu_ids.size());
    for (int i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipSetDevice(gpu_ids[i]));
        CUDA_ERROR(hipMemcpy(&truth[i], d_buf[i], sizeof(int),
                              hipMemcpyDeviceToHost));
    }

    // sync and check the output
    for (int i = 0; i < gpu_ids.size(); ++i) {
        int i_next = (i + 1) % gpu_ids.size();
        int i_prev = (i == 0) ? gpu_ids.size() - 1 : i - 1;

        printf("\n*** ID = %d", i);
        printf(
            "\n i_next = %d, correct_value= %d, value_from_device= %d, "
            "h_result= %d",
            i_next, values[i_next], truth[i_next], h_result[2 * i]);

        printf(
            "\n i_prev = %d, correct_value= %d, value_from_device= %d, "
            "h_result= %d\n",
            i_prev, values[i_prev], truth[i_prev], h_result[2 * i + 1]);
    }


    // clean up
    for (size_t i = 0; i < gpu_ids.size(); ++i) {
        CUDA_ERROR(hipGraphExecDestroy(exec_graphs[i]));
        CUDA_ERROR(hipGraphDestroy(graphs[i]));
        CUDA_ERROR(hipStreamDestroy(streams[i]));
        CUDA_ERROR(hipFree(d_buf[i]));
        CUDA_ERROR(hipEventDestroy(events[i]));
    }
}
